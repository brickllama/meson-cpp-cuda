#include "hip/hip_runtime.h"
#include "../include/cuda.cuh"
#include <stdio.h>
#include <string.h>

/**
 * @brief Says hello from the GPU.
 */
__global__ void HelloWorldKernel(char *gpuName)
{
    printf("%s says: Hello, World!\n", gpuName);
}

/**
 * @brief Tells the GPU to say hello.
 */
void HelloWorld()
{
    hipError_t error;

    /* Try to locate an eligible GPU. */
    int deviceNumber;
    error = hipGetDevice(&deviceNumber);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failure finding a NVIDIA GPU: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    /* Try to get the said GPU's device properties. */
    hipDeviceProp_t gpu;
    error = cudaGetDeviceProperties_v2(&gpu, deviceNumber);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failure getting gpu%d's device properties: %s\n", deviceNumber, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    /* Copy the gpu's name to the device. */
    char *gpuName;
    hipMalloc(&gpuName, sizeof(gpu.name));
    hipMemcpy(gpuName, gpu.name, sizeof(gpu.name), hipMemcpyHostToDevice);

    /* Say hello. */
    HelloWorldKernel<<<1, 1>>>(gpuName);
    hipDeviceSynchronize();

    /* Free the gpu's name. */
    hipFree(gpuName);
}